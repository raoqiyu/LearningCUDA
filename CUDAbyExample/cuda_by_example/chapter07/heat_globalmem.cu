#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "../common/cpu_anim.h"

#define DIM 512
#define PI 3.1415926535897932f
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED 0.25f

__global__ void copy_const_kernel(float *iptr, const float *cptr){
	// 2D grid, 2D block
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int offset = y * blockDim.x * gridDim.x + x;

	if(cptr[offset] != 0)
		iptr[offset] = cptr[offset];
}

__global__ void blend_kernel( float *outSrc, const float *inSrc){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int offset = y * blockDim.x*gridDim.x + x;

	int left = offset -1;
	int right = offset + 1;
	if (x == 0)  left++;
	if (right == DIM-1) right--;
	
	// the top and bottom cells are in the lines
	// above or below current line in a 2D rect
	int top = offset-DIM;
	int bottom = offset + DIM;
	if ( y == 0) top += DIM;
	if ( y == DIM-1) bottom -= DIM;

	outSrc[offset] = inSrc[offset] + SPEED*(inSrc[top] + inSrc[bottom]+
											inSrc[left]+ inSrc[right]-
											inSrc[offset]*4);
}
struct DataBlock{
	unsigned char	*output_bitmap;
	float			*dev_inSrc;
	float			*dev_outSrc;
	float			*dev_constSrc;
	CPUAnimBitmap	*bitmap;
	hipEvent_t		start, stop;
	float			totalTime;
	float			frames;
};

void anim_gpu(DataBlock *d, int ticks){
	HANDLE_ERROR( hipEventRecord( d->start, 0) );
	int threadsPerBlock=16;
	dim3 threads(threadsPerBlock, threadsPerBlock);
	dim3 blocks(DIM/threadsPerBlock, DIM/threadsPerBlock);
	CPUAnimBitmap *bitmap = d->bitmap;
	
	// 1、 保持有heater source的区域heat不变
	// 2、 根据公式更新grid
	// 3、 将本次更新后的grid作为下一次的输入
	for(int i=0; i < 90; i++){
		copy_const_kernel<<<blocks, threads>>>(d->dev_inSrc, d->dev_constSrc);
		blend_kernel<<<blocks, threads>>>(d->dev_outSrc, d->dev_inSrc);
		swap(d->dev_inSrc, d->dev_outSrc);
	}
	float_to_color<<<blocks, threads>>>(d->output_bitmap, d->dev_inSrc);

	HANDLE_ERROR( hipMemcpy(bitmap->get_ptr(), d->output_bitmap,
							 bitmap->image_size(), hipMemcpyDeviceToHost));

	HANDLE_ERROR( hipEventRecord( d->stop, 0));
	HANDLE_ERROR( hipEventSynchronize( d->stop));
	float elapsedTime;
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, d->start, d->stop));
	d->totalTime += elapsedTime;
	++d->frames;
	printf("Average Time per frame: %3.1f ms\n", d->totalTime/d->frames);
}

void anim_exit( DataBlock *d){
	hipFree( d->dev_inSrc);
	hipFree( d->dev_outSrc);
	hipFree( d->dev_constSrc);

	HANDLE_ERROR( hipEventDestroy( d->start));
	HANDLE_ERROR( hipEventDestroy( d->stop));
}


int main(void){
	DataBlock data;
	CPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;
	data.totalTime = 0;
	data.frames = 0;

	HANDLE_ERROR( hipEventCreate(&data.start));
	HANDLE_ERROR( hipEventCreate(&data.stop));

	// image has DIM*DIM cells, each cell's color represented by 4 chars (rgba)
	// bitmap.image_size() == DIM*DIM*4
	HANDLE_ERROR( hipMalloc( (void**) &data.output_bitmap, bitmap.image_size()));
	HANDLE_ERROR( hipMalloc( (void**) &data.dev_inSrc, bitmap.image_size()));
	HANDLE_ERROR( hipMalloc( (void**) &data.dev_outSrc, bitmap.image_size()));
	HANDLE_ERROR( hipMalloc( (void**) &data.dev_constSrc, bitmap.image_size()));


	float *temp = (float*) malloc(bitmap.image_size());
	for(int i = 0; i < DIM*DIM; i++){
		temp[i] = 0;
		int x = i % DIM;
		int y = i / DIM;
		if ((x>150) && (x<300) && (y>160) && (y<300))
			temp[i] = MAX_TEMP;
	}
	
	temp[DIM*50+50] = (MAX_TEMP + MIN_TEMP)/2;
    temp[DIM*350+50] = MIN_TEMP;
    temp[DIM*150+150] = MIN_TEMP;
    temp[DIM*100+350] = MIN_TEMP;
	for (int y=400; y<450; y++) {
		for (int x=200; x<250; x++) { 
			temp[x+y*DIM] = MIN_TEMP;
		}
	}
	
	HANDLE_ERROR( hipMemcpy( data.dev_constSrc, temp, bitmap.image_size(),
							hipMemcpyHostToDevice));

	for (int y=400; y<DIM; y++) { 
		for (int x=0; x<100; x++) {
        	temp[x+y*DIM] = MAX_TEMP;
         }
     }

	HANDLE_ERROR( hipMemcpy( data.dev_inSrc, temp,
                                 bitmap.image_size(),
                                 hipMemcpyHostToDevice ) );
	free( temp );
	
	bitmap.anim_and_exit( (void (*)(void*,int))anim_gpu,
					(void (*)(void*))anim_exit );
}

